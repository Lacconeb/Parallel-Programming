#include "hip/hip_runtime.h"
// Array multiplication: C = A * B:

// System includes
#include <stdio.h>
#include <assert.h>
#include <malloc.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"


#ifndef BLOCKSIZE
#define BLOCKSIZE		64		// number of threads per block
#endif

#ifndef SIZE
#define SIZE			512000	// array size
#endif

#ifndef NUMTRIALS
#define NUMTRIALS		100		// to make the timing more accurate
#endif

#ifndef TOLERANCE
#define TOLERANCE		0.00001f	// tolerance to relative error
#endif

// ranges for the random numbers:
const float XCMIN = 0.0;
const float XCMAX = 2.0;
const float YCMIN = 0.0;
const float YCMAX = 2.0;
const float RMIN = 0.5;
const float RMAX = 2.0;

// function prototypes:
float		Ranf(float, float);
int			Ranf(int, int);
void		TimeOfDaySeed();


// array multiplication (CUDA Kernel) on the device: C = A * B

__global__  void ArrayMul( float *A, float *B, float *C, float *D )
{
	
	
	unsigned int gid = blockIdx.x * blockDim.x + threadIdx.x;


	// randomize the location and radius of the circle:
	float xc = A[gid];
	float yc = B[gid];
	float  r = C[gid];

	// solve for the intersection using the quadratic formula:
	float a = 2.;
	float b = -2. * (xc + yc);
	float c = xc * xc + yc * yc - r * r;
	float d = b * b - 4. * a * c;

	if (d >= 0.) {
		// hits the circle:
		// get the first intersection:
		d = sqrt(d);
		float t1 = (-b + d) / (2. * a);	// time to intersect the circle
		float t2 = (-b - d) / (2. * a);	// time to intersect the circle
		float tmin = t1 < t2 ? t1 : t2;		// only care about the first intersection

		if (tmin >= 0.) {
			// where does it intersect the circle?
			float xcir = tmin;
			float ycir = tmin;

			// get the unitized normal vector at the point of intersection:
			float nx = xcir - xc;
			float ny = ycir - yc;
			float n = sqrt(nx * nx + ny * ny);
			nx /= n;	// unit vector
			ny /= n;	// unit vector

			// get the unitized incoming vector:
			float inx = xcir - 0.;
			float iny = ycir - 0.;
			float in = sqrt(inx * inx + iny * iny);
			inx /= in;	// unit vector
			iny /= in;	// unit vector

			// get the outgoing (bounced) vector:
			float dot = inx * nx + iny * ny;
			float outx = inx - 2. * nx * dot;	// angle of reflection = angle of incidence`
			float outy = iny - 2. * ny * dot;	// angle of reflection = angle of incidence`

			// find out if it hits the infinite plate:
			float t = (0. - ycir) / outy;

			if (t < 0.) {
				
			}
			else {
				D[gid] = 1;
			}
		}

		
	}

		
	

	/*
	__shared__ float prods[BLOCKSIZE];

	unsigned int numItems = blockDim.x;
	unsigned int tnum = threadIdx.x;
	unsigned int wgNum = blockIdx.x;
	unsigned int gid = blockIdx.x*blockDim.x + threadIdx.x;

	prods[tnum] = A[gid] * B[gid];

	for (int offset = 1; offset < numItems; offset *= 2)
	{
		int mask = 2 * offset - 1;
		__syncthreads();
		if ((tnum & mask) == 0)
		{
			prods[tnum] += prods[tnum + offset];
		}
	}

	__syncthreads();
	if (tnum == 0)
		C[wgNum] = prods[0];
	*/
}


// main program:

int
main( int argc, char* argv[ ] )
{
	int dev = findCudaDevice(argc, (const char **)argv);

	TimeOfDaySeed();

	// allocate host memory:

	float * xcs = new float [ SIZE ];
	float * ycs = new float [ SIZE ];
	float * rs = new float [ SIZE ];
	float * hD = new float [ SIZE ];

	for( int i = 0; i < SIZE; i++ )
	{
		xcs[i] = Ranf(XCMIN, XCMAX);
		ycs[i] = Ranf(YCMIN, YCMAX);
		rs[i] = Ranf(RMIN, RMAX);
		hD[i] = 0;
	}

	// allocate device memory:

	float *dA, *dB, *dC, *dD;

	dim3 dimsA( SIZE, 1, 1 );
	dim3 dimsB( SIZE, 1, 1 );
	dim3 dimsC( SIZE, 1, 1 );
	dim3 dimsD( SIZE, 1, 1 );

	//__shared__ float prods[SIZE/BLOCKSIZE];


	hipError_t status;
	status = hipMalloc( reinterpret_cast<void **>(&dA), SIZE*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&dB), SIZE*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&dC), SIZE*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc(reinterpret_cast<void**>(&dD), SIZE * sizeof(float));
		checkCudaErrors(status);


	// copy host memory to the device:

	status = hipMemcpy( dA, xcs, SIZE*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );
	status = hipMemcpy( dB, ycs, SIZE*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );
	status = hipMemcpy(dC, rs, SIZE * sizeof(float), hipMemcpyHostToDevice);
		checkCudaErrors(status);
	status = hipMemcpy(dD, hD, SIZE * sizeof(float), hipMemcpyHostToDevice);
		checkCudaErrors(status);

	// setup the execution parameters:

	dim3 threads(BLOCKSIZE, 1, 1 );
	dim3 grid( SIZE / threads.x, 1, 1 );

	// Create and start timer

	hipDeviceSynchronize( );

	// allocate CUDA events that we'll use for timing:

	hipEvent_t start, stop;
	status = hipEventCreate( &start );
		checkCudaErrors( status );
	status = hipEventCreate( &stop );
		checkCudaErrors( status );

	// record the start event:

	status = hipEventRecord( start, NULL );
		checkCudaErrors( status );

	// execute the kernel:

	for( int t = 0; t < NUMTRIALS; t++)
	{
	        ArrayMul<<< grid, threads >>>( dA, dB, dC, dD );
	}

	// record the stop event:

	status = hipEventRecord( stop, NULL );
		checkCudaErrors( status );

	// wait for the stop event to complete:

	status = hipEventSynchronize( stop );
		checkCudaErrors( status );

	float msecTotal = 0.0f;
	status = hipEventElapsedTime( &msecTotal, start, stop );
		checkCudaErrors( status );

	// compute and print the performance

	double secondsTotal = 0.001 * (double)msecTotal;
	double multsPerSecond = (float)SIZE * (float)NUMTRIALS / secondsTotal;
	double megaMultsPerSecond = multsPerSecond / 1000000.;
	fprintf( stderr, "Array Size = %10d, MegaMultReductions/Second = %10.2lf\n", SIZE, megaMultsPerSecond );

	// copy result from the device to the host:

	status = hipMemcpy( hD, dD, SIZE*sizeof(float), hipMemcpyDeviceToHost );
		checkCudaErrors( status );

	// check the sum :
	
	double sum = 0;
	for(int i = 0; i < SIZE; i++ )
	{
		sum += (double)hD[i];
	}
	float prob = (float)sum / (float)SIZE;

	fprintf(stderr, "\nsum = %lf\n", sum);
	fprintf( stderr, "\nprob = %lf\n", prob );
	

	// clean up memory:
	delete [ ] xcs;
	delete [ ] ycs;
	delete [ ] rs;
	delete [ ] hD;

	status = hipFree( dA );
		checkCudaErrors( status );
	status = hipFree( dB );
		checkCudaErrors( status );
	status = hipFree( dC );
		checkCudaErrors( status );
	status = hipFree( dD );
		checkCudaErrors(status);


	return 0;
}

float
Ranf(float low, float high)
{
	float r = (float)rand();               // 0 - RAND_MAX
	float t = r / (float)RAND_MAX;       // 0. - 1.

	return   low + t * (high - low);
}

int
Ranf(int ilow, int ihigh)
{
	float low = (float)ilow;
	float high = ceil((float)ihigh);

	return (int)Ranf(low, high);
}

void
TimeOfDaySeed()
{
	struct tm y2k = { 0 };
	y2k.tm_hour = 0;   y2k.tm_min = 0; y2k.tm_sec = 0;
	y2k.tm_year = 100; y2k.tm_mon = 0; y2k.tm_mday = 1;

	time_t  timer;
	time(&timer);
	double seconds = difftime(timer, mktime(&y2k));
	unsigned int seed = (unsigned int)(1000. * seconds);    // milliseconds
	srand(seed);
}